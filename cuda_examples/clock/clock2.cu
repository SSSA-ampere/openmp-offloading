#include "hip/hip_runtime.h"
/*
EXAMPLE SOURCE : 
https://forums.developer.nvidia.com/t/reading-globaltimer-register-or-calling-clock-clock64-in-loop-prevent-concurrent-kernel-execution/48600/8
https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#special-registers-clock64
https://docs.nvidia.com/cuda/inline-ptx-assembly/index.html

generating Asm PTX code
https://developer.nvidia.com/blog/cuda-pro-tip-view-assembly-code-correlation-nsight-visual-studio-edition/
https://stackoverflow.com/questions/20482686/how-to-get-the-assembly-code-of-a-cuda-kernel
$ nvcc -ptx -o kernel.ptx kernel.cu

.func  (.param .b64 func_retval0) clock64(

)
{
        .reg .b64       %rd<3>;


        // inline asm
        mov.u64         %rd1, %clock64;
        // inline asm
        mov.b64         %rd2, %rd1;
        st.param.b64    [func_retval0+0], %rd2;
        ret;
}





COMPILATION:
/usr/local/cuda-10.2/bin/nvcc -ccbin g++ -I../common/inc  -m64 -g -G    -gencode arch=compute_30,code=sm_30 -gencode arch=compute_30,code=compute_30 -o clock2  clock2.cu
/usr/local/cuda-10.2/bin/nvcc -ccbin g++ -I../common/inc  -m64 -g -G    -gencode arch=compute_72,code=sm_72 -gencode arch=compute_72,code=compute_72 -o clock2  clock2.cu

THIS EXAMPLE WORKS, WITH SOME LITTLE EXTRA TIME
*/
#include <stdio.h>

#define DELAY_VAL 5000000000ULL // about 5 secs
/*
long milliseconds()
{
    long            ms; // Milliseconds
    time_t          s;  // Seconds
    struct timespec spec;

    clock_gettime(CLOCK_MONOTONIC, &spec);

    s  = spec.tv_sec;
    ms = round(spec.tv_nsec / 1.0e6); // Convert nanoseconds to milliseconds
    return ms + s *1000;
}
*/

__global__ void child(){

    unsigned long long start = clock64();
    //for(long long int i=0;i<DELAY_VAL;i++);
    while (clock64()< start+DELAY_VAL);
    
}

int main(int argc, char* argv[]){

    hipStream_t st1, st2;
    hipStreamCreate(&st1);
    hipStreamCreate(&st2);

    clock_t ck_start = clock();
    //long start = milliseconds();
    //long now = 0;
    child<<<1,1,0,st1>>>();
    /*
    
        printf("running double kernel\n");
        while ( now < start + DELAY_VAL) {
            now = milliseconds();
        }
        printf("host finishing ...\n");
    */
    //parent<<<1,1,0,st2>>>();
    hipDeviceSynchronize();
    //printf ("Elapsed: %ld clicks.\n",milliseconds()-start);
    printf ("Kernel: %ld clicks.\n", clock()-ck_start);
    return 0;
}
